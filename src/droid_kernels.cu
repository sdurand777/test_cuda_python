#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Parallel.h>


__global__ void addition_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> a,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> b,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> c)
{
    // Calcul de l'index global du thread en 2D (x, y)
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Ligne du tenseur
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Colonne du tenseur

    // Vérification pour s'assurer que le thread est dans les limites du tenseur
    if (row < a.size(0) && col < a.size(1)) {
        // Addition des deux tenseurs élément par élément
        c[row][col] = a[row][col] + b[row][col];
    }
}


// Fonction pour imprimer un tenseur
void print_tensor(torch::Tensor tensor) {
    // Copier le tenseur du GPU vers le CPU
    auto cpu_tensor = tensor.to(torch::kCPU);
    auto data_ptr = cpu_tensor.data_ptr<float>();
    auto size = cpu_tensor.numel();
    
    std::cout << "Tensor values:\n";
    for (int i = 0; i < size; ++i) {
        if (i % cpu_tensor.size(1) == 0) std::cout << "\n";  // Nouvelle ligne pour chaque ligne du tenseur
        std::cout << data_ptr[i] << " ";
    }
    std::cout << std::endl;
}



torch::Tensor depth_filter_cuda(
        torch::Tensor a,
        torch::Tensor b)
{
    // Vérification que les tenseurs sont sur le GPU et ont le bon type de données
    TORCH_CHECK(a.device().is_cuda(), "Tensor a must be a CUDA tensor");
    TORCH_CHECK(b.device().is_cuda(), "Tensor b must be a CUDA tensor");
    TORCH_CHECK(a.scalar_type() == torch::kFloat32, "Tensor a must be of type float");
    TORCH_CHECK(b.scalar_type() == torch::kFloat32, "Tensor b must be of type float");

    // Vérification que les dimensions des tenseurs correspondent
    TORCH_CHECK(a.sizes() == b.sizes(), "Tensors a and b must have the same shape");

    // Créer un tenseur de sortie sur le GPU avec la même forme que a et b
    auto c = torch::zeros_like(a);

    //printf("Tensor a");
    //print_tensor(a);

    // Si vous avez besoin de transférer les tenseurs sur le CPU :
    torch::Tensor a_cpu = a.cpu();
    torch::Tensor b_cpu = b.cpu();

    // Obtenir des PackedTensorAccessor pour accéder aux données dans CUDA
    auto a_accessor = a.packed_accessor32<float,2,torch::RestrictPtrTraits>();
    auto b_accessor = b.packed_accessor32<float,2,torch::RestrictPtrTraits>();
    auto c_accessor = c.packed_accessor32<float,2,torch::RestrictPtrTraits>();

    // Configuration des dimensions des blocs et de la grille
    dim3 threadsPerBlock(16, 16);  // 16x16 threads par bloc
    dim3 numBlocks((a.size(1) + threadsPerBlock.x - 1) / threadsPerBlock.x,
            (a.size(0) + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Lancement du kernel CUDA
    addition_kernel<<<numBlocks, threadsPerBlock>>>(a_accessor, b_accessor, c_accessor);
    return c;
}


