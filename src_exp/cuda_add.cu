#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <iostream>

// Taille des tableaux (pour l'exemple)
#define N 1024

// Kernel CUDA pour l'addition de deux tableaux
__global__ void addition_kernel(const float* a, const float* b, float* c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

// Fonction exposée à Python pour l'addition des tableaux
void addition_cuda(pybind11::array_t<float> a, pybind11::array_t<float> b, pybind11::array_t<float> c) {
    // Accès aux données depuis les objets numpy
    auto buf_a = a.request();
    auto buf_b = b.request();
    auto buf_c = c.request();

    // Taille du tableau
    int size = buf_a.size;

    // Obtenir les pointeurs vers les données
    float* ptr_a = static_cast<float*>(buf_a.ptr);
    float* ptr_b = static_cast<float*>(buf_b.ptr);
    float* ptr_c = static_cast<float*>(buf_c.ptr);

    // Allocation sur le GPU
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size * sizeof(float));
    hipMalloc(&d_b, size * sizeof(float));
    hipMalloc(&d_c, size * sizeof(float));

    // Copier les données de l'hôte vers le GPU
    hipMemcpy(d_a, ptr_a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, ptr_b, size * sizeof(float), hipMemcpyHostToDevice);

    // Lancer le kernel CUDA
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    //addition_kernel<<<numBlocks, blockSize>>>(d_a, d_b, d_c, size);
    addition_kernel<<<1, 1>>>(d_a, d_b, d_c, size);

    // Copier le résultat du GPU vers l'hôte
    hipMemcpy(ptr_c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);

    // Libération de la mémoire GPU
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}


