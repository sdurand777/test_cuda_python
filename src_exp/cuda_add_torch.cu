#include "hip/hip_runtime.h"

#include <torch/extension.h>
#include <hip/hip_runtime.h>

// Taille des tableaux (pour l'exemple)
#define N 1024

// Kernel CUDA pour l'addition de deux tableaux
__global__ void addition_kernel(const float* a, const float* b, float* c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

// Fonction exposée à Python pour l'addition des tableaux
torch::Tensor addition_cuda(torch::Tensor a, torch::Tensor b) {
    // Vérifier que les tenseurs sont sur le GPU et de type float
    TORCH_CHECK(a.is_cuda(), "Le tenseur a doit être sur le GPU");
    TORCH_CHECK(b.is_cuda(), "Le tenseur b doit être sur le GPU");
    TORCH_CHECK(a.dtype() == torch::kFloat32, "Le tenseur a doit être de type float32");
    TORCH_CHECK(b.dtype() == torch::kFloat32, "Le tenseur b doit être de type float32");
    TORCH_CHECK(a.sizes() == b.sizes(), "Les deux tenseurs doivent avoir la même taille");

    // Créer un tenseur de sortie pour le résultat sur le GPU
    torch::Tensor c = torch::zeros_like(a);

    // Taille du tableau
    int size = a.numel();

    // Obtenir les pointeurs vers les données sur le GPU
    const float* d_a = a.data_ptr<float>();
    const float* d_b = b.data_ptr<float>();
    float* d_c = c.data_ptr<float>();

    // Lancer le kernel CUDA
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    addition_kernel<<<numBlocks, blockSize>>>(d_a, d_b, d_c, size);

    // Retourner le tenseur résultat
    return c;
}


